#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <stack>
#include <numeric>
#include <algorithm>
#include <bitset>
#include <cstdlib>

#include "parser.hpp"
#include <hip/hip_runtime.h>

int* d_constraints;
int* d_u;
size_t* d_offsets;
bool* d_changed;
bool* d_no_solution;
size_t* d_num_of_0s;

size_t N;
size_t unused_bits;
size_t total_elements;
size_t total_bytes;
size_t* offsets;
size_t* num_of_0s;
int* u;
char* singleton_domains;
char* domains;


typedef struct {
    size_t N;
    size_t total_bytes;
    
    char* domains;
    char* singleton_domains;
    char* checked_domains;
    size_t* cancelled_values;
    size_t* remaining_value;
} GPUNode;

void check_error(hipError_t err, const char* msg)
{
    if (err != hipSuccess)
    {
        std::cerr << msg << " : " << hipGetErrorString(err) << std::endl;
        std::cerr << hipGetErrorName(err) << std::endl;
        
        exit(EXIT_FAILURE);
    }
}

GPUNode* instantiate_gpu_node(const size_t& N, const size_t& total_bytes)
{
    hipError_t err;

    // Allocate GPUNode in unified memory (accessible by both host and device)
    GPUNode* node;
    err = hipMallocManaged(&node, sizeof(GPUNode)); check_error(err, "hipMallocManaged-Node");

    // Initialize const members
    node->N = N;
    node->total_bytes = total_bytes;

    // Allocate member arrays on the device
    err = hipMallocManaged(&node->domains, total_bytes); check_error(err, "hipMalloc-Domains");
    err = hipMallocManaged(&node->singleton_domains, (N + 7) / 8); check_error(err, "hipMalloc-SingletonDomains");
    err = hipMallocManaged(&node->checked_domains, (N + 7) / 8); check_error(err, "hipMalloc-CheckedDomains");
    err = hipMallocManaged(&node->cancelled_values, N * sizeof(size_t)); check_error(err, "hipMalloc-cancelledValues");
    err = hipMallocManaged(&node->remaining_value, N * sizeof(size_t)); check_error(err, "hipMalloc-RemainingValue");

    return node;
}

void instantiate_additional_data(Data& d)
{
    // Read the data from the Data object
    N = d.get_n();
    total_elements = std::accumulate(d.get_u(), d.get_u() + N, 0) + N;
    total_bytes = (total_elements + 7) / 8;
    offsets = new size_t[N];
    offsets[0] = 0;
    for (size_t i = 1; i < N; ++i)
    {
        offsets[i] = offsets[i-1] + d.get_u_at(i-1) + 1;
    }

    // Print the data
    std::cout << "N              : " << N << std::endl;
    std::cout << "Total elements : " << total_elements << std::endl;
    std::cout << "Total bytes    : " << total_bytes << std::endl;
    std::cout << "Offsets        : ";
    for (size_t i = 0; i < N; ++i)
    {
        std::cout << offsets[i] << " ";
    }
    std::cout << std::endl;

    // Linearize the constraint matrix
    int* constraints = new int[N * N];
    for (size_t i = 0; i < N; ++i)
    {
        for (size_t j = 0; j < N; ++j)
        {
            constraints[i * N + j] = d.get_C_at(i, j);
        }
    }

    // Allocate memory on the device
    hipError_t err;
    err = hipMalloc(&d_constraints, N * N * sizeof(int)); check_error(err, "hipMalloc-Constraints");
    err = hipMalloc(&d_u, N * sizeof(int)); check_error(err, "hipMalloc-U");
    err = hipMalloc(&d_offsets, N * sizeof(size_t)); check_error(err, "hipMalloc-Offsets");
    err = hipMalloc(&d_changed, sizeof(bool)); check_error(err, "hipMalloc-Changed");
    err = hipMalloc(&d_no_solution, sizeof(bool)); check_error(err, "hipMalloc-NoSolution");
    err = hipMallocManaged(&d_num_of_0s, sizeof(size_t) * N); check_error(err, "hipMalloc-AllSingletons");
    
    // Copy the data to the device
    err = hipMemcpy(d_constraints, constraints, N * N * sizeof(int), hipMemcpyHostToDevice); check_error(err, "hipMemcpy-Constraints");
    err = hipMemcpy(d_u, d.get_u(), N * sizeof(int), hipMemcpyHostToDevice); check_error(err, "hipMemcpy-U");
    err = hipMemcpy(d_offsets, offsets, N * sizeof(size_t), hipMemcpyHostToDevice); check_error(err, "hipMemcpy-Offsets");

} 

void free_gpu_node(GPUNode* node)
{
    hipError_t err;

    // Free member arrays
    err = hipFree(node->domains); check_error(err, "hipFree-Domains");
    err = hipFree(node->singleton_domains); check_error(err, "hipFree-SingletonDomains");
    err = hipFree(node->checked_domains); check_error(err, "hipFree-CheckedDomains");
    err = hipFree(node->cancelled_values); check_error(err, "hipFree-cancelledValues");
    err = hipFree(node->remaining_value); check_error(err, "hipFree-RemainingValue");

    // Free the GPUNode structure itself
    err = hipFree(node); check_error(err, "hipFree-Node");
}

void free_additional_data()
{
    hipError_t err;

    // Free the memory on the device
    err = hipFree(d_constraints); check_error(err, "hipFree-Constraints");
    err = hipFree(d_u); check_error(err, "hipFree-U");
    err = hipFree(d_offsets); check_error(err, "hipFree-Offsets");
    err = hipFree(d_changed); check_error(err, "hipFree-Changed");
    err = hipFree(d_no_solution); check_error(err, "hipFree-NoSolution");
    err = hipFree(d_num_of_0s); check_error(err, "hipFree-AllSingletons");
}

void copy_data_from_parent_to_child(GPUNode* parent, GPUNode* child)
{
    hipError_t err;

    // Copy the data from the source to the destination
    child->N = parent->N;
    child->total_bytes = parent->total_bytes;

    err = hipMemcpy(child->domains, parent->domains, parent->total_bytes, hipMemcpyDeviceToDevice); check_error(err, "hipMemcpy-DeviceToDevice-Domains");
    err = hipMemcpy(child->singleton_domains, parent->singleton_domains, (parent->N + 7) / 8, hipMemcpyDeviceToDevice); check_error(err, "hipMemcpy-DeviceToDevice-SingletonDomains");
    err = hipMemcpy(child->checked_domains, parent->checked_domains, (parent->N + 7) / 8, hipMemcpyDeviceToDevice); check_error(err, "hipMemcpy-DeviceToDevice-CheckedDomains");
    err = hipMemcpy(child->cancelled_values, parent->cancelled_values, parent->N * sizeof(size_t), hipMemcpyDeviceToDevice); check_error(err, "hipMemcpy-DeviceToDevice-cancelledValues");
    err = hipMemcpy(child->remaining_value, parent->remaining_value, parent->N * sizeof(size_t), hipMemcpyDeviceToDevice); check_error(err, "hipMemcpy-DeviceToDevice-RemainingValue");
}

__global__ void fixpoint_kernel(GPUNode* current, size_t* d_offsets, int* d_constraints, int* d_u)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= current->N) return;

    // Check if a domain is a singleton
    if (current->singleton_domains[idx / 8] & (1 << (idx % 8)))
    {
        const int value = current->remaining_value[idx];
        // If the domain is a singleton, we need to apply the constraints
        for (size_t i = 0; i < current->N; ++i)
        {   
            if (idx == i) continue;

            if (d_constraints[idx * current->N + i] == 1)
            {
                // If the value is inside the domain of the other variable
                if (value <= d_u[i])
                {
                    const size_t byte_idx = (d_offsets[i] + value) / 8;
                    const size_t bit_idx = (d_offsets[i] + value) % 8;
                    if (!(current->domains[byte_idx] & (1 << bit_idx)))
                    {
                        current->domains[byte_idx] |= (1 << bit_idx);
                    }
                }
            }
        }
    }
}


__global__ void check_singleton_domains(GPUNode* current, size_t* num_of_0s)
{
    // Map a thread to each byte of the current->domains array
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= current->total_bytes) return;

    // Count the number of 0s in the byte
    char byte = current->domains[idx];
    int count = 0;
    for (int i = 0; i < 8; ++i)
    {
        if (!(byte & (1 << i)))
        {
            count++;
        }
    }
    num_of_0s[idx] = count;
}

__global__ void count_num_of_0s(char* current, size_t total_bytes, size_t* num_of_0s)
{
    // Map a thread to each byte of the current->domains array
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_bytes) return;

    // Count the number of 0s in the byte
    char byte = current[idx];
    int count = 0;
    for (int i = 0; i < 8; ++i)
    {
        if (!(byte & (1 << i)))
        {
            count++;
        }
    }
    num_of_0s[idx] = count;
}


void fixpoint(GPUNode* current, std::stack<GPUNode*>& stack, size_t& exploredTree, size_t& exploredSol)
{
    hipError_t err;
    bool changed = true;
    bool no_solution = false;


    while (changed)
    {
        changed = false;
        err = hipMemcpy(d_changed, &changed, sizeof(bool), hipMemcpyHostToDevice); check_error(err, "hipMemcpy-HostToDevice-Changed");
        fixpoint_kernel<<<(current->N + 31) / 32, 32>>>(current, d_offsets, d_constraints, d_u);
        err = hipDeviceSynchronize(); check_error(err, "hipDeviceSynchronize");
        err = hipGetLastError(); check_error(err, "fixpoint_kernel");

        count_num_of_0s<<<(current->total_bytes + 31) / 32, 32>>>(current->domains, current->total_bytes, d_num_of_0s);
        err = hipDeviceSynchronize(); check_error(err, "hipDeviceSynchronize");
        err = hipGetLastError(); check_error(err, "count_num_of_0s");
        
        const size_t num_of_0s = std::accumulate(d_num_of_0s, d_num_of_0s + current->total_bytes, 0);
        if (num_of_0s == current->N + unused_bits)
        {
            exploredSol++;
            return;
        }
        if (num_of_0s < current->N + unused_bits)
        {
            return;
        }

        // Now iterate over the domains that are not singletons
        for (size_t domain = 0; domain < current->N; ++domain)
        {
            if (!(current->singleton_domains[domain / 8] & (1 << (domain % 8))))
            {
                // Count all the 0s in the domain
                size_t count = 0;
                for (size_t i = 0; i <= u[domain]; ++i)
                {
                    const size_t byte_idx = (offsets[domain] + i) / 8;
                    const size_t bit_idx = (offsets[domain] + i) % 8;
                    if (!(current->domains[byte_idx] & (1 << bit_idx)))
                    {
                        count++;
                    }
                }
                if (count == 1)
                {
                    // Set the domain as singleton
                    current->singleton_domains[domain / 8] |= (1 << (domain % 8));
                    // Find the value that is 0
                    size_t value = 0;
                    for (size_t i = 0; i <= u[domain]; ++i)
                    {
                        const size_t byte_idx = (offsets[domain] + i) / 8;
                        const size_t bit_idx = (offsets[domain] + i) % 8;
                        if (!(current->domains[byte_idx] & (1 << bit_idx)))
                        {
                            value = i;
                            break;
                        }
                    }
                    current->remaining_value[domain] = value;
                    current->cancelled_values[domain] = u[domain];
                    changed = true;
                }   
            }
        }

    }


    if (std::accumulate(d_num_of_0s, d_num_of_0s + current->total_bytes, 0) == N + unused_bits)
    {
        if(exploredSol%1000 == 0)
        {
            std::cout << "Explored tree  : " << exploredSol << std::endl;
        }
        exploredSol++;
        return;
    }

    // Else, we need to select the first non-singleton domain
    // Copy the singleton domains from the device to the host

    err = hipMemcpy(singleton_domains, current->singleton_domains, (N+7)/8, hipMemcpyDeviceToHost); check_error(err, "hipMemcpy-DeviceToHost-SingletonDomains");


    // Find the first non-singleton domain
    size_t branching_variable = 0;
    for (branching_variable = 0; branching_variable < N; ++branching_variable)
    {
        if (!(singleton_domains[branching_variable / 8] & (1 << (branching_variable % 8))))
        {
            break;
        }
    }

    // Actually branch on the variable
    err = hipMemcpy(domains, current->domains, current->total_bytes, hipMemcpyDeviceToHost); check_error(err, "hipMemcpy-DeviceToHost-Domains");
    for (int branch_value = 0; branch_value <= u[branching_variable]; ++branch_value)
    {
        const size_t byte_idx = (offsets[branching_variable] + branch_value) / 8;
        const size_t bit_idx = (offsets[branching_variable] + branch_value) % 8;
        // Branch if the variable has a 0
        if (!(domains[byte_idx] & (1 << bit_idx)))
        {
            exploredTree++; 
            // std::cout << "Branching on value " << branch_value << std::endl;
            GPUNode* child = instantiate_gpu_node(N, total_bytes);
            // Copy the data from the parent to the child
            copy_data_from_parent_to_child(current, child);
            // Set the domain as a singleton
            child->singleton_domains[branching_variable / 8] |= (1 << (branching_variable % 8));
            // Set the remaining value
            child->remaining_value[branching_variable] = branch_value;
            // Cancel all the other values
            for (int i = 0; i <= u[branching_variable]; ++i)
            {
                if (i != branch_value)
                {
                    const size_t byte_idx = (offsets[branching_variable] + i) / 8;
                    const size_t bit_idx = (offsets[branching_variable] + i) % 8;
                    child->domains[byte_idx] |= (1 << bit_idx);
                }
            }
            
            stack.push(child);
        }
    }
}


void test_gpu_fixpoint(char* str)
{
    Data d;
    d.read_input(str);

    instantiate_additional_data(d);

    GPUNode* root = instantiate_gpu_node(N, total_bytes);

    num_of_0s = new size_t[total_bytes];
    singleton_domains = new char[(N+7)/8];
    domains = new char[total_bytes];
    unused_bits = 8 * total_bytes - total_elements;

    u = d.get_u();
    std::stack<GPUNode*> stack;
    stack.push(root);

    size_t exploredTree = 0;
    size_t exploredSol = 0;

    std::cout << "Starting the fixpoint algorithm" << std::endl;
    auto start = std::chrono::high_resolution_clock::now();
    while (!stack.empty())
    {
        GPUNode* current = stack.top();
        stack.pop();

        fixpoint(current, stack, exploredTree, exploredSol);

        free_gpu_node(current);
    }

    auto end = std::chrono::high_resolution_clock::now();

    free_additional_data();
    std::cout << "===================================" << std::endl;
    std::cout << "Explored tree  : " << exploredTree << std::endl;
    std::cout << "Explored sol   : " << exploredSol << std::endl;
    std::cout << "Time           : " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;

}


int main(int argc, char** argv)
{  
    test_gpu_fixpoint(argv[1]);
    
    return 0;
}
